#include "hip/hip_runtime.h"
#include <unistd.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <math.h>
#include <time.h>

#include "variable_types.h"
#include "global.h"
#include "disk_profile.h"
#include "ppm.h"
#include "output.h"
#include "timestep.h"
#include "FARGO.h"
#include "cuSafe.cu"

using namespace std;

//=======================================================================
// CPU Grid
//=======================================================================

sdp *zxa;
sdp *zxc;
sdp *zdx;
sdp *zya;
sdp *zyc;
sdp *zdy;
sdp *zza;
sdp *zzc;
sdp *zdz;

#include "init.cpp"

void copy_bfile(ifstream &bfile, GPU_plan *set)
{
  for (int n=0; n<nDev; n++)
    bfile.read((char*)set[n].h_rings, set[n].memsize);
  return;
}

void show_init(sdp B0, double ttau)
{
  sdp *beta = new sdp[imax*jmax*kmax];
  SymDisk *val = new SymDisk[imax*kmax];

  zxa=new sdp[imax];
  zxc=new sdp[imax];
  zdx=new sdp[imax];

  zya=new sdp[jmax];
  zyc=new sdp[jmax];
  zdy=new sdp[jmax];

  zza=new sdp[kmax];
  zzc=new sdp[kmax];
  zdz=new sdp[kmax];

//-----------------------------------------------------------------------------------------
// Generate Grid
    init_grid();

//-----------------------------------------------------------------------------------------
// Initializing
  init_den(val);

  init_speed(val);

  double temp;
  int ind;
  double m_dot = -1.5*ss_alpha*sc_h*sc_h*twopi*sqrt(twopi)*sc_h;

  for (int i=0; i<imax; i++)
  {
    temp=0;
    for (int k=0; k<kmax; k++)
    {
      ind = i+imax*k;
      temp += val[ind].r*val[ind].u*twopi*zxc[i]*zdz[k];
    }
    cout << zxc[i] << " " << val[i+imax*9].r << " " << val[i].v << " " << val[i].u << " " << temp/m_dot << " " << val[ind].r*val[ind].u*twopi*zxc[i]/(-1.5*ss_alpha*sc_h*sc_h*twopi) << endl;
  }
  wait_f_r();
  for (int k=0; k<kmax; k++)
  {
     cout << k << " " << zza[k] << " " << zdz[k] << endl;
  }
  wait_f_r();
  for (int j=0; j<jmax; j++)
  {
     cout << j << " " << zya[j] << " " << zdy[j] << endl;
  }
  wait_f_r();
  for (int i=0; i<imax; i++)
  {
     cout << i << " " << zxa[i] << " " << zdx[i] << endl;
  }
/*
  for (int i=0; i<imax; i++)
  {
     cout << i << " " << zxa[i] << " " << zdx[i]/zxc[i] << " " << val[i+imax*(kmax-1)].u << endl;
  }
*/
  cout << (0.5*zxc[0]*zyc[0]/(val[imax*(kmax-1)].v+sqrt(get_cs2(zxc[0]*sin(zzc[kmax-1])))-zxc[0]))/(twopi*5.0) << endl;
  cout << val[imax*(kmax-1)].v << " " << sqrt(get_cs2(zxc[0]*sin(zzc[kmax-1]))) << " " << sqrt(val[imax*(kmax-1)].p/val[imax*(kmax-1)].r) << " " << zxc[0]*sin(zzc[kmax-1]) << endl;

  delete[] beta, val;
  return;
}

void kernel(bool restart, int startat, int morph, string morph_path, sdp B0, double ttau)
{
  string mainp = path_to_cwd();

  string label = create_label();
  string ifname, pfname, bfname, tfname;
  ofstream ifile, pfile, bfile, tfile;

  ifname  = mainp+"/files/initial_"+label+".dat";
  tfname  = mainp+"/files/time_"+label+".dat";
  pfname  = mainp+"/read/para_"+label+".dat";
  bfname  = mainp+"/binary/binary_"+label+"_";

  #if file_flag == 1
  ofstream result_file;
  string result_fname;
  result_fname  = mainp+"/files/result_"+label;
  #endif

  if (ndim==1 && jmax*kmax!=1){
    cout << "Requesting 1D problem but arrays are dimensioned for 2/3D" << endl;
    return;
  }
  if (ndim==2 && kmax!=1){
    cout << "Requesting 2D problem but arrays are dimensioned for 3D" << endl;
    return;
  }
  if (ndim==3 && kmax==1){
    cout << "Requesting 3D problem but arrays are dimensioned for 2D" << endl;
    return;
  }
//-----------------------------------------------------------------------------------------
// Begin by allocating device memory

  GPU_plan *set = new GPU_plan[nDev];

  //if (nDev==2 && startid==0) {set[0].id=0; set[1].id=2;}
  //else
  for (int n=0; n<nDev; n++) set[n].id = startid+n;
  for (int n=0; n<nDev; n++) set[n].N_ring = imax/nDev;
  for (int n=0; n<imax%nDev; n++) set[nDev-1-n].N_ring++;
  for (int n=0; n<nDev; n++) set[n].N_ring *= kmax;

  //P2P_all_enable(set);

  for (int n=0; n<nDev; n++)
  {
    CudaSafeCall( hipSetDevice(set[n].id) );
    CudaSafeCall( hipDeviceSetCacheConfig(hipFuncCachePreferShared) );
    CudaSafeCall( hipStreamCreate(&set[n].stream) );
    CudaSafeCall( hipEventCreate(&set[n].event) );
   
    printf("Device %i contains %i rings.\n", set[n].id, set[n].N_ring);
    set[n].memsize = set[n].N_ring*sizeof(hydr_ring);

    if (n==0) set[n].istart = 0;
    else      set[n].istart = set[n-1].istart + set[n-1].iblk;
    set[n].iblk = set[n].N_ring/kmax;
    set[n].kstart = 0;
    set[n].kblk = kmax;
    
    set[n].sx_grid.x = jmax;
    set[n].sx_grid.y = set[n].kblk;
    set[n].sx_grid.z = 1;

    //printf(" %i, %i, %i \n", set[n].sx_grid.x, set[n].sx_grid.y, set[n].sx_grid.z );

    set[n].sy_grid.x = set[n].iblk;
    set[n].sy_grid.y = set[n].kblk;
    set[n].sy_grid.z = 1;

    //printf(" %i, %i, %i \n", set[n].sy_grid.x, set[n].sy_grid.y, set[n].sy_grid.z );

    set[n].sz_grid.x = set[n].iblk;
    set[n].sz_grid.y = jmax;
    set[n].sz_grid.z = (set[n].kblk/realarr) + (bool)(set[n].kblk%realarr);

    //printf(" %i, %i, %i \n", set[n].sz_grid.x, set[n].sz_grid.y, set[n].sz_grid.z );

    set[n].t_grid.x = set[n].iblk;
    set[n].t_grid.y = set[n].kblk;
    set[n].t_grid.z = 1;

    printf("Device %i: Allocating %i MB for grid ...", set[n].id, set[n].memsize/1048576);
    CudaSafeCall( hipMalloc( (void**)&set[n].rings, set[n].memsize ) );
    CudaSafeCall( hipHostMalloc( (void**)&set[n].h_rings, set[n].memsize ) );
    printf(" done\n");

    #if ndim==3
    printf("Device %i: Allocating %i MB for boundaries ...", set[n].id, (2*n_pad*imax*sizeof(hydr_ring)+4*kmax*sizeof(hydr_ring))/1048576);
    CudaSafeCall( hipMalloc( (void**)&set[n].udr, n_pad*imax*sizeof(hydr_ring) ) );
    CudaSafeCall( hipMalloc( (void**)&set[n].top, n_pad*imax*sizeof(hydr_ring) ) );
    #else
    printf("Device %i: Allocating %i MB for boundaries ...", set[n].id, (4*kmax*sizeof(hydr_ring))/1048576);
    #endif
    CudaSafeCall( hipMalloc( (void**)&set[n].lft, n_pad*kmax*sizeof(hydr_ring) ) );
    CudaSafeCall( hipMalloc( (void**)&set[n].rgh, n_pad*kmax*sizeof(hydr_ring) ) );
    CudaSafeCall( hipMalloc( (void**)&set[n].cp_lft, n_pad*kmax*sizeof(hydr_ring) ) );
    CudaSafeCall( hipMalloc( (void**)&set[n].cp_rgh, n_pad*kmax*sizeof(hydr_ring) ) );
    printf(" done\n");

    #if file_flag==1
    CudaSafeCall( hipMalloc( (void**)&set[n].d_output1, sizeof(sdp)*jmax*set[n].iblk*set[n].kblk ) );
    CudaSafeCall( hipHostMalloc( (void**)&set[n].h_output1, sizeof(sdp)*jmax*set[n].iblk*set[n].kblk ) );

    CudaSafeCall( hipMalloc( (void**)&set[n].d_output2, sizeof(sdp)*jmax*set[n].iblk*set[n].kblk ) );
    CudaSafeCall( hipHostMalloc( (void**)&set[n].h_output2, sizeof(sdp)*jmax*set[n].iblk*set[n].kblk ) );

    CudaSafeCall( hipMalloc( (void**)&set[n].d_output3, sizeof(sdp)*jmax*set[n].iblk*set[n].kblk ) );
    CudaSafeCall( hipHostMalloc( (void**)&set[n].h_output3, sizeof(sdp)*jmax*set[n].iblk*set[n].kblk ) );

    CudaSafeCall( hipMalloc( (void**)&set[n].d_output4, sizeof(sdp)*jmax*set[n].iblk*set[n].kblk ) );
    CudaSafeCall( hipHostMalloc( (void**)&set[n].h_output4, sizeof(sdp)*jmax*set[n].iblk*set[n].kblk ) );

    CudaSafeCall( hipMalloc( (void**)&set[n].d_output5, sizeof(sdp)*jmax*set[n].iblk*set[n].kblk ) );
    CudaSafeCall( hipHostMalloc( (void**)&set[n].h_output5, sizeof(sdp)*jmax*set[n].iblk*set[n].kblk ) );

    cout << "  Allocated " << 5*sizeof(sdp)*imax*jmax*kmax/1024/1024/nDev << "MB for output" << endl;
    #endif

    CudaSafeCall( hipMalloc( (void**)&set[n].val, imax*kmax*sizeof(SymDisk) ) );

    CudaSafeCall( hipMalloc( (void**)&set[n].dt, sizeof(sdp) ) );
    CudaSafeCall( hipHostMalloc( (void**)&set[n].h_dt, sizeof(sdp) ) );
    CudaSafeCall( hipHostMalloc( (void**)&set[n].dt_1D, sizeof(sdp)*set[n].iblk ) );
    CudaSafeCall( hipHostMalloc( (void**)&set[n].dt_2D, sizeof(sdp)*set[n].iblk*set[n].iblk ) );


    CudaSafeCall( hipMalloc( (void**)&set[n].d_output, sizeof(sdp) ) );
    CudaSafeCall( hipHostMalloc( (void**)&set[n].h_output, sizeof(sdp) ) );
    cout << endl;
  }

  hydr_ring *lft = new hydr_ring[n_pad*kmax];
  hydr_ring *rgh = new hydr_ring[n_pad*kmax];
  hydr_ring *udr = new hydr_ring[n_pad*imax];
  hydr_ring *top = new hydr_ring[n_pad*imax];
  SymDisk *val = new SymDisk[imax*kmax];

  zxa=new sdp[imax];
  zxc=new sdp[imax];
  zdx=new sdp[imax];

  zya=new sdp[jmax];
  zyc=new sdp[jmax];
  zdy=new sdp[jmax];

  zza=new sdp[kmax];
  zzc=new sdp[kmax];
  zdz=new sdp[kmax];

  syncdevices(set);

//-----------------------------------------------------------------------------------------
// Generate Grid

  int npic = startat;

  if (restart)
  {
    ifstream sfile;
    open_binary_file(sfile, bfname+frame_num(npic));
    if(!sfile)
    {
      cout << endl << " Looking for " << bfname+frame_num(npic) << endl;
      cout << endl << " Restart data does not exist." << endl;
      return;
    }
    else
    {
      cout << endl << " Restarting from t=" << tmovie*npic << endl;
    }
    copy_bfile(sfile, set);
    copy_grid(set);
  }
  else 
  {
    init_grid();
  }

//-----------------------------------------------------------------------------------------
// Initializing
  sdp dt = endtime;

  sdp simtime = tmovie*(double)startat;
  sdp timep   = 0.0;
  sdp timem   = 0.0;
  int ncycle  = 0;
  int ncycp   = 0;

  init_den(val);
  cout << " density is set" << endl;

  init_speed(val);
  cout << " speed is set" << endl;

  if (!restart) init_cells(set, val);
  init_bound(lft, rgh, udr, top);
  cout << " boundary is set" << endl;

  //add_atmosphere(set);

  for (int n=0; n<nDev; n++)
  {
    CudaSafeCall( hipMemcpyAsync( set[n].rings, set[n].h_rings, set[n].memsize, hipMemcpyHostToDevice ) );

    CudaSafeCall( hipMemcpyAsync( set[n].lft, lft, n_pad*kmax*sizeof(hydr_ring), hipMemcpyHostToDevice ) );
    CudaSafeCall( hipMemcpyAsync( set[n].rgh, rgh, n_pad*kmax*sizeof(hydr_ring), hipMemcpyHostToDevice ) );

    #if ndim == 3
    CudaSafeCall( hipMemcpyAsync( set[n].udr, udr, n_pad*imax*sizeof(hydr_ring), hipMemcpyHostToDevice ) );
    CudaSafeCall( hipMemcpyAsync( set[n].top, top, n_pad*imax*sizeof(hydr_ring), hipMemcpyHostToDevice ) );
    #endif

    CudaSafeCall( hipMemcpyAsync( set[n].val, val, imax*kmax*sizeof(SymDisk), hipMemcpyHostToDevice ) );
  }
  syncdevices(set);

//-----------------------------------------------------------------------------------------
  #if dump_flag==1
  if (!restart)
  {
    open_output_file(pfile, pfname);
    write_para_file(pfile, mainp, label);
    pfile.close();

    open_binary_file(bfile, bfname+frame_num(npic));
    save_cells(bfile, set);
    bfile.close();
  }
  open_output_file(ifile, ifname);
  write_initial_file(ifile, zxa, zdx);
  ifile.close();

  if (!restart) open_output_file(tfile, tfname);
  else        append_output_file(tfile, tfname);
  #endif

  #if file_flag==1
  open_binary_file(result_file, result_fname);
  for (int n=0; n<nDev; n++)
  {
    CudaSafeCall( hipSetDevice(set[n].id) );
    clear_output<<< kmax , imax, 0 , set[n].stream >>>(set[n].d_output1, set[n].d_output2, set[n].d_output3, set[n].d_output4, set[n].d_output5);
  }

  result *a = new result;
  (*a).q_ratio = M_p;
  (*a).aspect_ratio = sc_h;
  (*a).alpha = ss_alpha;
  #endif

  body planet;
  planet.m = M_p;
  planet.x = R_p;
  planet.y = pi;
  planet.vx = 0.0;
  planet.vy = pow(planet.x,-1.5)-FrRot;

/*
  //for (int i=0; i<n_pad; i++)   printf("%f : %.12f \n", lft[i+n_pad*0].xc, lft[i+n_pad*0].r[20] );
  //for (int i=0; i<imax; i++)   printf("%f : %.12f \n", set[0].h_rings[i+imax*0].xc, set[0].h_rings[i+imax*0].r[20] );
  //for (int i=0; i<n_pad; i++)   printf("%f : %.12f \n", rgh[i+n_pad*0].xc, rgh[i+n_pad*0].r[20] );
  //wait_f_r();
  
  sdp tmp;
  if (ndim == 3) tmp = (ymax-ymin) * (cos(zmin)-cos(zmax)) * third*(pow(xmax,3)-pow(xmin,3));
  else           tmp = (ymax-ymin) * 0.5*(pow(xmax,2)-pow(xmin,2));
  printf("True grid volume: %.12f \n", tmp );

  tmp=0.0;
  for (int i=0; i<imax; i++)
  {
    for (int k=0; k<kmax; k++)
    {
      for (int j=0; j<jmax; j++)
      {
        if (ndim == 3) tmp += (cos(zza[k])-cos(zza[k]+zdz[k])) * zdx[i]*(zxa[i]*(zxa[i] + zdx[i]) + zdx[i]*zdx[i]*third) * zdy[j];
        else           tmp += set[0].h_rings[i].xvol * set[0].h_rings[i].yvol[j];//zdx[i]*(zxa[i] + zdx[i]*0.5) * zdy[j];
      }
    }
  }
  printf("CPU volume sum: %.12f \n", tmp );

  tmp = GPU_output_reduction(set, planet, 0.0);
  printf("GPU volume sum: %.12f \n", tmp );


  dt = courant * (twopi/(sdp)jmax) / (sc_h/zxc[0] + pow(zxc[0],-1.5) - pow(0.75,-1.5));
  printf("Analytic dt with/o FARGO: %.12f \n", dt );

  dt=1.0;
  for (int i=0; i<imax; i++)
  {
    for (int k=0; k<kmax; k++)
    {
      for (int j=0; j<jmax; j++)
      {
        if (ndim == 3) dt = min(dt, courant*zxc[i]*sin(zzc[k])*zdy[j]/(sc_h) );
        else           dt = min(dt, courant*zxc[i]*zdy[j]/(sc_h) );
        dt = min(dt, courant*zdx[i]/(sc_h + val[i+imax*k].u) );
        if (ndim == 3) dt = min(dt, courant*zxc[i]*zdz[k]/(sc_h + val[i+imax*k].w) );
      }
    }
  }
  printf("CPU dt with FARGO: %.12f \n", dt );
*/
//============================================================================================
//                         MAIN COMPUTATIONAL LOOP

  clock_t begin, elapse;
  double speed;
  bool make_pict = false;
  bool prin_info = false;
  bool cont_simu = true;
  sdp old_dt;
  begin = clock();

  while (cont_simu)
  {
    dt = get_dt(dt, set);
    Index_Shift(set, dt);
    planet.m = set_M_p(simtime);
   
    if ( ncycle == ncycend )
    {
      prin_info = true;
      cont_simu = false;
    }
    if ( simtime + dt >= endtime )  // set dt to land on endtime
    {
      cout << "cutting to the end..." << " " << ncycle << " " << ncycend << endl;
      old_dt = dt;
      dt = endtime - simtime;
      prin_info = true;
      make_pict = true;
      cont_simu = false;
    }
    if ( timem + dt >= tmovie ) // set dt to land on tmovie
    {
      old_dt = dt;
      dt = tmovie - timem;
      make_pict = true;
    }

    #if ndim == 2
    bundle_sweep2D(set, dt, planet);
    #elif ndim == 3
    bundle_sweep3D(set, dt, planet);
    #endif
    planet.y += dt*(1.0-FrRot);
    if (planet.y<0.0)    planet.y += twopi;
    if (planet.y>=twopi) planet.y -= twopi;

    #if file_flag==1
    for (int n=0; n<nDev; n++) 
    {
      CudaSafeCall( hipSetDevice(set[n].id) );
      cal_output<<< set[n].s_grid , idim, 0 , set[n].stream >>>(set[n].cells, dt, set[n].d_output1, set[n].d_output2, set[n].d_output3, set[n].d_output4, set[n].d_output5);
    }
    #endif

    simtime += dt;
    timem   += dt;
    ncycle++;
    ncycp++;

    if ( ncycp == nprin )
    {
      ncycp = 0;
      prin_info = true;
    }

    if (make_pict)
    {
      #if dump_flag == 1
      for (int n=0; n<nDev; n++)
      {
        CudaSafeCall( hipMemcpy( set[n].h_rings, set[n].rings, set[n].memsize, hipMemcpyDeviceToHost ) );
      }
      npic++;
      open_binary_file(bfile, bfname+frame_num(npic));
      save_cells(bfile, set);
      bfile.close();
      cout << " check point " << npic << " saved at " << bfname+frame_num(npic) << endl;
      #endif

      timem = 0.0;
      make_pict = false;
      dt = old_dt;
    }

    if (prin_info)
    {
      #if dump_flag==1
      tfile << simtime << " " << GPU_output_reduction(set, planet, 0.0)*sc_h*sc_h/M_p/M_p << endl;
      #endif

      elapse = clock()-begin;
      speed = (double)elapse/(double)CLOCKS_PER_SEC/(double)ncycle;
      cout << endl;
      cout << "# of steps = " << ncycle << " ( t/T = "<< simtime/endtime << ", dt = " << dt << " )" << endl;
      cout << "Average Speed is " << speed << " seconds per time step." << endl;
      cout << "Estimated time for completion: " << (speed*(double)ncycle)/60.0 << " of "
           << ((endtime-tmovie*(double)startat)*speed*(double)ncycle/(simtime-tmovie*(double)startat))/60.0 << " minutes." << endl;
      cout << endl;
      prin_info = false;
    }
  }
/*
  for (int n=0; n<nDev; n++)
  {
    CudaSafeCall( hipMemcpy( set[n].h_rings, set[n].rings, set[n].memsize, hipMemcpyDeviceToHost ) );
  }

  for (int i=0; i<n_pad; i++)  printf("%f : %.12e \n", lft[i].xc, lft[i].r[20]*cpow(lft[i].xc,p_alpha)-1.0 );
  for (int i=0; i<imax; i++)   printf("%f : %.12e \n", set[0].h_rings[i].xc, set[0].h_rings[i].r[20]*cpow(set[0].h_rings[i].xc,p_alpha)-1.0 );
  for (int i=0; i<n_pad; i++)  printf("%f : %.12e \n", rgh[i].xc, rgh[i].r[20]*cpow(rgh[i].xc,p_alpha)-1.0 );
  wait_f_r();
*/
  //P2P_all_disable(set);

  #if file_flag == 1
  for (int n=0; n<nDev; n++)
  {
    CudaSafeCall( hipSetDevice(set[n].id) );
    CudaSafeCall( hipMemcpy( set[n].h_output1, set[n].d_output1, sizeof(sdp)*imax*jmax*kmax/nDev, hipMemcpyDeviceToHost ) );
    CudaSafeCall( hipMemcpy( set[n].h_output2, set[n].d_output2, sizeof(sdp)*imax*jmax*kmax/nDev, hipMemcpyDeviceToHost ) );
    CudaSafeCall( hipMemcpy( set[n].h_output3, set[n].d_output3, sizeof(sdp)*imax*jmax*kmax/nDev, hipMemcpyDeviceToHost ) );
    CudaSafeCall( hipMemcpy( set[n].h_output4, set[n].d_output4, sizeof(sdp)*imax*jmax*kmax/nDev, hipMemcpyDeviceToHost ) );
    CudaSafeCall( hipMemcpy( set[n].h_output5, set[n].d_output5, sizeof(sdp)*imax*jmax*kmax/nDev, hipMemcpyDeviceToHost ) );
  }

  int n;
  int ind;
  (*a).start_t = tmovie*(double)startat;
  (*a).end_t = simtime;

  for (int i=0; i<imax; i++)
  {
    (*a).x[i]  = zxc[i];
    (*a).dx[i] = zdx[i];
  }

  for (int j=0; j<jmax; j++)
  {
    (*a).y[j]  = zyc[j];
    (*a).dy[j] = zdy[j];
  }

  for (int k=0; k<kmax; k++)
  {
    (*a).z[k]  = zzc[k];
    (*a).dz[k] = zdz[k];
  }

  for (int i=0; i<imax; i++)
    for (int j=0; j<jmax; j++)
    {
      n = j*nDev/jmax;
      for (int k=0; k<kmax; k++)
      {
        ind = i+imax*((j-n*jmax/nDev)+(jmax/nDev)*k);
        (*a).r[i][j][k] = set[n].h_output1[ind]/(simtime - tmovie*(double)startat);
        (*a).p[i][j][k] = set[n].h_output2[ind]/set[n].h_output1[ind];
        (*a).u[i][j][k] = set[n].h_output3[ind]/set[n].h_output1[ind];
        (*a).v[i][j][k] = set[n].h_output4[ind]/set[n].h_output1[ind];
        (*a).w[i][j][k] = set[n].h_output5[ind]/set[n].h_output1[ind];
      }
    }
  result_file.write((char*)a, sizeof(result));
  result_file.close();
  #endif
 
  for (int n=0; n<nDev; n++)
  {
    CudaSafeCall( hipSetDevice(set[n].id) );
    CudaSafeCall( hipFree( set[n].rings ) );
    CudaSafeCall( hipHostFree( set[n].h_rings ) );

    CudaSafeCall( hipFree( set[n].lft ) );
    CudaSafeCall( hipFree( set[n].rgh ) );
    CudaSafeCall( hipFree( set[n].cp_lft ) );
    CudaSafeCall( hipFree( set[n].cp_rgh ) );
    #if ndim==3
    CudaSafeCall( hipFree( set[n].udr ) );
    CudaSafeCall( hipFree( set[n].top ) );
    #endif

    #if file_flag==1
    CudaSafeCall( hipFree( set[n].d_output1 ) );
    CudaSafeCall( hipHostFree( set[n].h_output1 ) );

    CudaSafeCall( hipFree( set[n].d_output2 ) );
    CudaSafeCall( hipHostFree( set[n].h_output2 ) );

    CudaSafeCall( hipFree( set[n].d_output3 ) );
    CudaSafeCall( hipHostFree( set[n].h_output3 ) );

    CudaSafeCall( hipFree( set[n].d_output4 ) );
    CudaSafeCall( hipHostFree( set[n].h_output4 ) );

    CudaSafeCall( hipFree( set[n].d_output5 ) );
    CudaSafeCall( hipHostFree( set[n].h_output5 ) );
    #endif

    CudaSafeCall( hipFree( set[n].val ) );

    CudaSafeCall( hipFree( set[n].dt ) );
    CudaSafeCall( hipHostFree( set[n].h_dt ) );

    CudaSafeCall( hipFree( set[n].d_output ) );
    CudaSafeCall( hipHostFree( set[n].h_output ) );
    CudaSafeCall( hipDeviceReset() );
  }
  delete[] lft,rgh,udr,top,val;
  #if file_flag == 1
  delete[] a;
  #endif
//                           END OF MAIN LOOP
//============================================================================================
  return;
}

int main(int narg, char *args[])
{
  //signal(SIGINT,int_handler);
  double ttau = 1.0;
  sdp B0 = 0.2;

  #if opac_flag == 0
  B0 = 0.0;
  #endif

  bool restart = false;
  int morph = 0;
  string morph_path;
  
  if      (narg==1)
  {
    restart = false;
    kernel(restart, 0, 0, morph_path, B0, ttau);
  }
  else if (narg==2)
  {
    if (string(args[1])=="init")
    {
      show_init(B0, ttau);
    }
    else
    {
      restart = true;
      kernel(restart, atof(args[1]), 0, morph_path, B0, ttau);
    }
  }
  else if (narg==3) 
  {
    restart = false; 
    morph = atof(args[2]); 
    morph_path = string(args[1]);
    kernel(restart, 0, morph, morph_path, B0, ttau);
  }     

  return 0;
}
